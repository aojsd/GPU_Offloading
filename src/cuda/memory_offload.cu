#include "hip/hip_runtime.h"
/**
 * @file memory_offload.cu
 * @brief Empirically tests three strategies for overlapping data movement with computation
 * for a large matrix multiplication problem (A * B = C) where matrix A is partially
 * offloaded (e.g., Host DRAM, PCIe, NVLink, C2C).
 *
 * Shapes: A[N x H], B[H x S], C[N x S]
 *
 * This version uses a CUDA Graph-based approach for all tests and includes
 * a fast, GPU-based verification function.
 *
 * Case 1: Explicit Overlap
 * - Uses a CUDA Graph to model the dependencies.
 * - Subcase 1 (default): Offloads to pinned host memory, transfers over PCIe (H2D).
 * - Subcase 2 (--nvlink): Offloads to a peer GPU's memory, transfers over NVLink (D2D).
 *
 * Case 2: Overlap with UVM Prefetch
 * - Matrices are allocated with `hipMallocManaged`.
 * - An async prefetch of the offloaded data is launched concurrently
 * with the main compute kernel, allowing the driver to overlap the transfer and
 * computation.
 *
 * Case 3: Legacy Zero-Copy (Interleaved Unified Kernel)
 * - A single, robust kernel is launched that is aware of both device VRAM and
 * pinned host RAM (zero-copy) for matrix A.
 * - The kernel re-orders logical work in a user-configurable "X-to-1" start/end
 * pattern to force a mix of high- and low-latency memory requests on the SMs,
 * enabling the GPU's latency-hiding capabilities.
 *
 * ---
 *
 * Compilation using Makefile:
 * make
 * (Optional: specify GPU architecture, e.g., `make ARCH=sm_90a`)
 *
 * Compilation (manual):
 * nvcc -O3 -arch=sm_90a memory_offload.cu -o memory_offload
 * (Adjust -arch=sm_XX to your GPU's compute capability)
 *
 * ---
 *
 * Usage:
 * ./memory_offload [options]
 * Run with -h or --help for a full list of options.
 */
#include <iostream>
#include <vector>
#include <string>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <iomanip>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CHECK_CUDA(call)                                                 \
    do                                                                   \
    {                                                                    \
        hipError_t err = call;                                          \
        if (err != hipSuccess)                                          \
        {                                                                \
            fprintf(stderr, "CUDA Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", err,               \
                    hipGetErrorString(err));                            \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    } while (0)

// Forward declaration for the main test function
void runBandwidthExtensionTest(int N, int H, int S, float offload_ratio, int trials, int device_id, int interleave_ratio);

// Kernel configuration
constexpr int TILE_DIM = 32;
// Maximum grid dimension size for Y and Z axes on modern GPUs
constexpr unsigned int MAX_GRID_DIM = 65535;

// Helper function to calculate a 2D grid that can handle large row counts
dim3 calculate_grid_dims(int num_elements_x, int num_elements_y) {
    long long total_blocks_y = (num_elements_y + TILE_DIM - 1) / TILE_DIM;
    long long grid_y = std::min((long long)MAX_GRID_DIM, total_blocks_y);
    long long grid_z = (total_blocks_y + grid_y - 1) / grid_y;

    if (grid_z > MAX_GRID_DIM) {
        fprintf(stderr, "Error: Matrix dimensions exceed launch capabilities.\n");
        exit(EXIT_FAILURE);
    }

    return dim3((num_elements_x + TILE_DIM - 1) / TILE_DIM, (unsigned int)grid_y, (unsigned int)grid_z);
}

// Tiled Matrix Multiplication Kernel (for separate launches and verification)
__global__ void matMulKernel(float *C, const float *A, const float *B, int N, int H, int S, int startRow, int numRows)
{
    long long total_row_blocks = (numRows + TILE_DIM - 1) / TILE_DIM;
    long long linear_block_idx_y = (long long)blockIdx.z * gridDim.y + blockIdx.y;

    if (linear_block_idx_y >= total_row_blocks) return;

    __shared__ float sA[TILE_DIM][TILE_DIM];
    __shared__ float sB[TILE_DIM][TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = linear_block_idx_y * TILE_DIM + ty + startRow;
    int col = blockIdx.x * TILE_DIM + tx;

    float C_val = 0.0f;
    
    for (int t = 0; t < (H + TILE_DIM - 1) / TILE_DIM; ++t)
    {
        long long local_row = (long long)row - startRow;
        if (row < (startRow + numRows) && (t * TILE_DIM + tx) < H) {
            sA[ty][tx] = A[local_row * H + t * TILE_DIM + tx];
        } else {
            sA[ty][tx] = 0.0f;
        }

        if (col < S && (t * TILE_DIM + ty) < H) {
            sB[ty][tx] = B[(long long)(t * TILE_DIM + ty) * S + col];
        } else {
            sB[ty][tx] = 0.0f;
        }
        
        __syncthreads();

        for (int k = 0; k < TILE_DIM; ++k) C_val += sA[ty][k] * sB[k][tx];
        __syncthreads();
    }

    if (row < (startRow + numRows) && col < S) C[(long long)row * S + col] = C_val;
}

// **REWRITTEN:** This is the correct implementation of your "X-to-1" design.
// It computes C[i] using A[i], but the order in which the 'i's are processed is interleaved.
__global__ void interleavedScratchpadMatMulKernel(
    float *C, 
    const float *A_resident, 
    const float *A_offload, 
    const float *B, 
    int N, int H, int S, 
    int N_resident,
    int interleave_ratio)
{
    long long total_tile_rows = (N + TILE_DIM - 1) / TILE_DIM;
    long long logical_tile_idx = (long long)blockIdx.z * gridDim.y + blockIdx.y;

    if (logical_tile_idx >= total_tile_rows) return;

    __shared__ float sA[TILE_DIM][TILE_DIM];
    __shared__ float sB[TILE_DIM][TILE_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // **KEY LOGIC:** The block computes the result for its assigned LOGICAL tile.
    int logical_start_row = logical_tile_idx * TILE_DIM;
    int tile_start_col = blockIdx.x * TILE_DIM;

    float C_val = 0.0f;
    for (int t = 0; t < (H + TILE_DIM - 1) / TILE_DIM; ++t)
    {
        // Load the tile for matrix A into shared memory (sA) row by row.
        int row_to_load = logical_start_row + ty;
        int col_to_load = t * TILE_DIM + tx;

        // Use a simple check against N_resident to determine the memory source for the LOGICAL row.
        if (row_to_load < N && col_to_load < H) {
            if (row_to_load < N_resident) {
                // This row is in the resident (VRAM) buffer.
                sA[ty][tx] = A_resident[ (long long)row_to_load * H + col_to_load ];
            } else {
                // This row is in the offloaded (Zero-Copy) buffer.
                long long offload_row_index = (long long)row_to_load - N_resident;
                sA[ty][tx] = A_offload[ offload_row_index * H + col_to_load ];
            }
        } else {
            sA[ty][tx] = 0.0f;
        }
        
        // Load tile for B into shared memory (sB) - always from VRAM.
        int b_row = t * TILE_DIM + ty;
        if (b_row < H && tile_start_col + tx < S) {
            sB[ty][tx] = B[ (long long)b_row * S + (tile_start_col + tx) ];
        } else {
            sB[ty][tx] = 0.0f;
        }
        
        __syncthreads();

        for (int k = 0; k < TILE_DIM; ++k) C_val += sA[ty][k] * sB[k][tx];
        __syncthreads();
    }

    // Write final result to C using the LOGICAL tile index to ensure correct, contiguous output.
    int final_row = logical_start_row + ty;
    int final_col = tile_start_col + tx;
    if (final_row < N && final_col < S) {
        C[(long long)final_row * S + final_col] = C_val;
    }
}


// New kernel to compare two matrices on the GPU and find the maximum absolute error
__global__ void compareAndFindMaxErrorKernel(const float* C_original, const float* C_verify, float* d_max_error, size_t total_elements) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_elements; i += gridDim.x * blockDim.x) {
        float error = fabsf(C_original[i] - C_verify[i]);
        
        float current_max = *d_max_error;
        while (error > current_max) {
            float previous_max = __uint_as_float(atomicCAS((unsigned int*)d_max_error, __float_as_uint(current_max), __float_as_uint(error)));
            if (previous_max == current_max) break;
            current_max = previous_max;
        }
    }
}

// Kernel to initialize the cuRAND states
__global__ void setupCurandKernel(hiprandState_t *states, unsigned long long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &states[id]);
}

// Kernel to initialize a matrix with random values
__global__ void initMatrixGpuKernel(float* matrix, size_t num_elements, hiprandState_t* states) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState_t local_state = states[id];
    for (size_t i = 0; i < num_elements; i += stride) {
        matrix[i] = hiprand_uniform(&local_state);
    }
    states[id] = local_state;
}

// Host function to orchestrate matrix initialization on the GPU
void init_matrices_on_gpu(float* d_A, float* d_B, int N, int H, int S) {
    std::cout << "Initializing matrices on GPU... " << std::flush;
    
    size_t num_elements_A = (size_t)N * H;
    if (d_A == nullptr) num_elements_A = 0;

    size_t num_elements_B = (size_t)H * S;
    if (d_B == nullptr) num_elements_B = 0;

    if (num_elements_A == 0 && num_elements_B == 0) {
        std::cout << "Skipped (no matrices to init).\n";
        return;
    }

    size_t max_elements = std::max(num_elements_A, num_elements_B);

    int threads_per_block = 256;
    int blocks = std::min(16384, (int)((max_elements + threads_per_block - 1) / threads_per_block));
    size_t num_states = (size_t)threads_per_block * blocks;

    hiprandState_t* d_rand_states;
    CHECK_CUDA(hipMalloc(&d_rand_states, num_states * sizeof(hiprandState_t)));
    setupCurandKernel<<<blocks, threads_per_block>>>(d_rand_states, time(NULL));

    if (num_elements_A > 0) initMatrixGpuKernel<<<blocks, threads_per_block>>>(d_A, num_elements_A, d_rand_states);
    if (num_elements_B > 0) initMatrixGpuKernel<<<blocks, threads_per_block>>>(d_B, num_elements_B, d_rand_states);
    
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(d_rand_states));
    std::cout << "Done.\n";
}

// New GPU-based verification function
void verify_result_gpu(const float* d_A, const float* d_B, const float* d_C_original, int N, int H, int S) {
    std::cout << "\nVerifying result on GPU... " << std::flush;
    
    size_t C_size = (size_t)N * S * sizeof(float);
    float* d_C_verify;
    CHECK_CUDA(hipMalloc(&d_C_verify, C_size));

    dim3 threadsPerBlock(TILE_DIM, TILE_DIM);
    dim3 blocksPerGrid = calculate_grid_dims(S, N);
    matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_C_verify, d_A, d_B, N, H, S, 0, N);

    float* d_max_error;
    float h_max_error = 0.0f;
    CHECK_CUDA(hipMalloc(&d_max_error, sizeof(float)));
    CHECK_CUDA(hipMemset(d_max_error, 0, sizeof(float)));

    int compare_threads = 256;
    int compare_blocks = std::min(1024, (int)(((size_t)N * S + compare_threads - 1) / compare_threads));
    compareAndFindMaxErrorKernel<<<compare_blocks, compare_threads>>>(d_C_original, d_C_verify, d_max_error, (size_t)N * S);
    
    CHECK_CUDA(hipMemcpy(&h_max_error, d_max_error, sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());

    std::cout << "Done.\nMaximum absolute error: " << h_max_error << std::endl;

    CHECK_CUDA(hipFree(d_C_verify));
    CHECK_CUDA(hipFree(d_max_error));
}


// **RESTORED:** This function handles the "fast path" case where offload_ratio is 0 for non-extension tests.
void runSingleKernelTest(int N, int H, int S, int trials, bool use_uvm, int device_id) {
    std::cout << "Zero offload ratio detected. Running simplified single-kernel test (CUDA Graph Version).\n";

    size_t A_size = (size_t)N * H * sizeof(float);
    size_t B_size = (size_t)H * S * sizeof(float);
    size_t C_size = (size_t)N * S * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    if (use_uvm) {
        CHECK_CUDA(hipMallocManaged(&d_A, A_size));
        CHECK_CUDA(hipMallocManaged(&d_B, B_size));
        CHECK_CUDA(hipMallocManaged(&d_C, C_size));
        init_matrices_on_gpu(d_A, d_B, N, H, S);

        CHECK_CUDA(hipMemAdvise(d_A, A_size, hipMemAdviseSetPreferredLocation, device_id));
        CHECK_CUDA(hipMemAdvise(d_B, B_size, hipMemAdviseSetPreferredLocation, device_id));
        CHECK_CUDA(hipMemAdvise(d_C, C_size, hipMemAdviseSetPreferredLocation, device_id));
        CHECK_CUDA(hipMemPrefetchAsync(d_A, A_size, device_id, stream));
        CHECK_CUDA(hipMemPrefetchAsync(d_B, B_size, device_id, stream));
        CHECK_CUDA(hipMemPrefetchAsync(d_C, C_size, device_id, stream));
        CHECK_CUDA(hipStreamSynchronize(stream));
    } else {
        CHECK_CUDA(hipMalloc(&d_A, A_size));
        CHECK_CUDA(hipMalloc(&d_B, B_size));
        CHECK_CUDA(hipMalloc(&d_C, C_size));
        init_matrices_on_gpu(d_A, d_B, N, H, S);
    }

    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    CHECK_CUDA(hipGraphCreate(&graph, 0));

    hipKernelNodeParams kernel_params = {0};
    kernel_params.func = (void*)matMulKernel;
    kernel_params.gridDim = calculate_grid_dims(S, N);
    kernel_params.blockDim = dim3(TILE_DIM, TILE_DIM);
    void *kernel_args[] = {&d_C, &d_A, &d_B, &N, &H, &S, new int(0), new int(N)};
    kernel_params.kernelParams = kernel_args;
    
    hipGraphNode_t start_node, stop_node, kernel_node;
    CHECK_CUDA(hipGraphAddEventRecordNode(&start_node, graph, nullptr, 0, start));
    CHECK_CUDA(hipGraphAddKernelNode(&kernel_node, graph, &start_node, 1, &kernel_params));
    CHECK_CUDA(hipGraphAddEventRecordNode(&stop_node, graph, &kernel_node, 1, stop));
    
    CHECK_CUDA(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

    const int WARMUP_COUNT = 5;
    std::cout << "Performing " << WARMUP_COUNT << " warm-up runs... " << std::flush;
    for (int i = 0; i < WARMUP_COUNT; ++i) CHECK_CUDA(hipGraphLaunch(graph_exec, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));
    std::cout << "Done.\n";

    std::vector<float> kernel_times;
    for (int i = 0; i < trials; ++i) {
        CHECK_CUDA(hipGraphLaunch(graph_exec, stream));
        CHECK_CUDA(hipStreamSynchronize(stream));
        
        float ms;
        CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));
        kernel_times.push_back(ms);
    }

    auto avg = [](const std::vector<float>& v) { return std::accumulate(v.begin(), v.end(), 0.0f) / v.size(); };
    double avg_time_ms = avg(kernel_times);
    double effective_bandwidth = (A_size + B_size) / (avg_time_ms * 1e6);

    std::cout << "\n--- Timings (avg over " << trials << " trials) ---\n";
    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Effective Bandwidth (GB/s): " << std::setw(8) << effective_bandwidth << "\n";
    std::cout << "Total Kernel Time:          " << std::setw(8) << avg_time_ms << " ms\n";

    verify_result_gpu(d_A, d_B, d_C, N, H, S);
    
    delete (int*)kernel_args[6];
    delete (int*)kernel_args[7];
    CHECK_CUDA(hipGraphExecDestroy(graph_exec));
    CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipStreamDestroy(stream));
}


void runExplicitOverlapTest(int N, int H, int S, float offload_ratio, int trials, bool use_nvlink, int device_id) {
    if (use_nvlink) {
        std::cout << "\n--- Running Case 1: Explicit Overlap with NVLink (D2D) ---\n";
    } else {
        std::cout << "\n--- Running Case 1: Explicit Overlap with PCIe (H2D) ---\n";
    }

    if (offload_ratio == 0.0f) {
        runSingleKernelTest(N, H, S, trials, false, device_id);
        return;
    }

    size_t A_size = (size_t)N * H * sizeof(float);
    size_t B_size = (size_t)H * S * sizeof(float);
    size_t C_size = (size_t)N * S * sizeof(float);
    int N_offload = static_cast<int>(N * offload_ratio);
    int N_resident = N - N_offload;
    size_t A_offload_size = (size_t)N_offload * H * sizeof(float);
    
    std::cout << "Resident Rows: " << N_resident << ", Offloaded Rows: " << N_offload << std::endl;

    float* h_A_pinned_offload = nullptr;
    float* d_A_peer_offload = nullptr;
    hipMemcpyKind copyKind;
    int peerDeviceId = -1;

    if (use_nvlink) {
        int device_count;
        CHECK_CUDA(hipGetDeviceCount(&device_count));
        peerDeviceId = (device_id + 1) % device_count;

        int canAccessPeer;
        CHECK_CUDA(hipDeviceCanAccessPeer(&canAccessPeer, device_id, peerDeviceId));
        if (canAccessPeer) {
            std::cout << "Enabling peer access from Device " << device_id << " to Device " << peerDeviceId << std::endl;
            CHECK_CUDA(hipSetDevice(device_id));
            CHECK_CUDA(hipDeviceEnablePeerAccess(peerDeviceId, 0));
        } else {
            std::cerr << "Error: Peer access between Device " << device_id << " and " << peerDeviceId << " is not supported." << std::endl;
            exit(EXIT_FAILURE);
        }
        
        CHECK_CUDA(hipSetDevice(peerDeviceId));
        CHECK_CUDA(hipMalloc(&d_A_peer_offload, A_offload_size));
        CHECK_CUDA(hipSetDevice(device_id));

        copyKind = hipMemcpyDeviceToDevice;
    } else {
        CHECK_CUDA(hipHostAlloc(&h_A_pinned_offload, A_offload_size, hipHostMallocDefault));
        copyKind = hipMemcpyHostToDevice;
    }

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, A_size));
    CHECK_CUDA(hipMalloc(&d_B, B_size));
    CHECK_CUDA(hipMalloc(&d_C, C_size));

    init_matrices_on_gpu(d_A, d_B, N, H, S);

    if (use_nvlink) {
        CHECK_CUDA(hipMemcpy(d_A_peer_offload, d_A + (size_t)N_resident * H, A_offload_size, hipMemcpyDeviceToDevice));
    } else {
        CHECK_CUDA(hipMemcpy(h_A_pinned_offload, d_A + (size_t)N_resident * H, A_offload_size, hipMemcpyDeviceToHost));
    }
    
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    CHECK_CUDA(hipGraphCreate(&graph, 0));

    hipEvent_t start, stop, transferStart, transferStop, compute1Start, compute1Stop, compute2Start, compute2Stop;
    CHECK_CUDA(hipEventCreate(&start)); CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventCreate(&transferStart)); CHECK_CUDA(hipEventCreate(&transferStop));
    CHECK_CUDA(hipEventCreate(&compute1Start)); CHECK_CUDA(hipEventCreate(&compute1Stop));
    CHECK_CUDA(hipEventCreate(&compute2Start)); CHECK_CUDA(hipEventCreate(&compute2Stop));

    hipGraphNode_t start_node, stop_node, memcpy_node, kernel1_node, kernel2_node, sync_node;
    hipGraphNode_t event_transfer_start, event_transfer_stop, event_compute1_start, event_compute1_stop, event_compute2_start, event_compute2_stop;

    CHECK_CUDA(hipGraphAddEventRecordNode(&start_node, graph, nullptr, 0, start));

    void* memcpy_src = use_nvlink ? (void*)d_A_peer_offload : (void*)h_A_pinned_offload;
    CHECK_CUDA(hipGraphAddEventRecordNode(&event_transfer_start, graph, &start_node, 1, transferStart));
    CHECK_CUDA(hipGraphAddMemcpyNode1D(&memcpy_node, graph, &event_transfer_start, 1, d_A + (size_t)N_resident * H, memcpy_src, A_offload_size, copyKind));
    CHECK_CUDA(hipGraphAddEventRecordNode(&event_transfer_stop, graph, &memcpy_node, 1, transferStop));

    hipKernelNodeParams kernel1_params = {0};
    kernel1_params.func = (void*)matMulKernel;
    kernel1_params.gridDim = calculate_grid_dims(S, N_resident);
    kernel1_params.blockDim = dim3(TILE_DIM, TILE_DIM);
    void *kernel1_args[] = {&d_C, &d_A, &d_B, &N, &H, &S, new int(0), new int(N_resident)};
    kernel1_params.kernelParams = kernel1_args;
    
    CHECK_CUDA(hipGraphAddEventRecordNode(&event_compute1_start, graph, &start_node, 1, compute1Start));
    CHECK_CUDA(hipGraphAddKernelNode(&kernel1_node, graph, &event_compute1_start, 1, &kernel1_params));
    CHECK_CUDA(hipGraphAddEventRecordNode(&event_compute1_stop, graph, &kernel1_node, 1, compute1Stop));

    hipGraphNode_t sync_deps[] = {event_transfer_stop, event_compute1_stop};
    CHECK_CUDA(hipGraphAddEmptyNode(&sync_node, graph, sync_deps, 2));

    hipKernelNodeParams kernel2_params = {0};
    kernel2_params.func = (void*)matMulKernel;
    kernel2_params.gridDim = calculate_grid_dims(S, N_offload);
    kernel2_params.blockDim = dim3(TILE_DIM, TILE_DIM);
    void *kernel2_args[] = {&d_C, &d_A, &d_B, &N, &H, &S, new int(N_resident), new int(N_offload)};
    kernel2_params.kernelParams = kernel2_args;

    CHECK_CUDA(hipGraphAddEventRecordNode(&event_compute2_start, graph, &sync_node, 1, compute2Start));
    CHECK_CUDA(hipGraphAddKernelNode(&kernel2_node, graph, &event_compute2_start, 1, &kernel2_params));
    CHECK_CUDA(hipGraphAddEventRecordNode(&event_compute2_stop, graph, &kernel2_node, 1, compute2Stop));
    CHECK_CUDA(hipGraphAddEventRecordNode(&stop_node, graph, &event_compute2_stop, 1, stop));
    
    CHECK_CUDA(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

    const int WARMUP_COUNT = 5;
    std::cout << "Performing " << WARMUP_COUNT << " warm-up runs... " << std::flush;
    for (int i = 0; i < WARMUP_COUNT; ++i) {
        if (!use_nvlink) {
            CHECK_CUDA(hipMemcpy(h_A_pinned_offload, d_A + (size_t)N_resident * H, A_offload_size, hipMemcpyDeviceToHost));
        }
        CHECK_CUDA(hipGraphLaunch(graph_exec, stream));
    }
    CHECK_CUDA(hipStreamSynchronize(stream));
    std::cout << "Done.\n";

    std::vector<double> total_times, transfer_times, compute1_times, compute2_times;
    for (int i = 0; i < trials; ++i) {
        if (!use_nvlink) {
            CHECK_CUDA(hipMemcpy(h_A_pinned_offload, d_A + (size_t)N_resident * H, A_offload_size, hipMemcpyDeviceToHost));
        }
        
        CHECK_CUDA(hipGraphLaunch(graph_exec, stream));
        CHECK_CUDA(hipStreamSynchronize(stream));

        float ms_total, ms_transfer, ms_compute1, ms_compute2;
        CHECK_CUDA(hipEventElapsedTime(&ms_total, start, stop));
        CHECK_CUDA(hipEventElapsedTime(&ms_transfer, transferStart, transferStop));
        CHECK_CUDA(hipEventElapsedTime(&ms_compute1, compute1Start, compute1Stop));
        CHECK_CUDA(hipEventElapsedTime(&ms_compute2, compute2Start, compute2Stop));
        total_times.push_back(ms_total);
        transfer_times.push_back(ms_transfer);
        compute1_times.push_back(ms_compute1);
        compute2_times.push_back(ms_compute2);
    }
    
    auto avg = [](const std::vector<double>& v) { return std::accumulate(v.begin(), v.end(), 0.0f) / v.size(); };
    std::string bw_label = use_nvlink ? "NVLink Transfer (D2D):" : "PCIe Transfer (H2D): ";

    std::cout << "\n--- Timings (avg over " << trials << " trials) ---\n";
    std::cout << std::fixed << std::setprecision(3);
    std::cout << bw_label << std::setw(8) << avg(transfer_times) << " ms\n";
    std::cout << "Compute (Resident Data):  " << std::setw(8) << avg(compute1_times) << " ms\n";
    std::cout << "Compute (Offloaded Data): " << std::setw(8) << avg(compute2_times) << " ms\n";
    std::cout << "--------------------------------------\n";
    std::string bw_rate_label = use_nvlink ? "NVLink Bandwidth (GB/s): " : "PCIe Bandwidth (GB/s): ";
    std::cout << bw_rate_label << std::setw(8) << (A_offload_size / (1e6 * avg(transfer_times))) << " GB/s\n";
    double total_compute_time = avg(compute1_times) + avg(compute2_times);
    std::cout << "GPU Throughput (GB/s): " << std::setw(8) << ((A_size + B_size) / (1e6 * total_compute_time)) << " GB/s\n";
    std::cout << "--------------------------------------\n";
    std::cout << "Total Kernel Time:    " << std::setw(8) << avg(total_times) << " ms\n";
    std::cout << "Total Compute Time = " << total_compute_time << " ms\n";
    
    verify_result_gpu(d_A, d_B, d_C, N, H, S);
    
    if (use_nvlink) {
        CHECK_CUDA(hipFree(d_A_peer_offload));
        CHECK_CUDA(hipDeviceDisablePeerAccess(peerDeviceId));
    } else {
        CHECK_CUDA(hipHostFree(h_A_pinned_offload));
    }
    delete (int*)kernel1_args[6]; delete (int*)kernel1_args[7];
    delete (int*)kernel2_args[6]; delete (int*)kernel2_args[7];
    CHECK_CUDA(hipGraphExecDestroy(graph_exec)); CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipEventDestroy(start)); CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipEventDestroy(transferStart)); CHECK_CUDA(hipEventDestroy(transferStop));
    CHECK_CUDA(hipEventDestroy(compute1Start)); CHECK_CUDA(hipEventDestroy(compute1Stop));
    CHECK_CUDA(hipEventDestroy(compute2Start)); CHECK_CUDA(hipEventDestroy(compute2Stop));
    CHECK_CUDA(hipFree(d_A)); CHECK_CUDA(hipFree(d_B)); CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipStreamDestroy(stream));
}

void runUvmTest(int N, int H, int S, float offload_ratio, int trials, int device_id) {
    const bool SERIALIZE_KERNELS = true;

    if (SERIALIZE_KERNELS) {
        std::cout << "\n--- Running Case 2: UVM with Serial Kernels (Multi-Stream) ---\n";
    } else {
        std::cout << "\n--- Running Case 2: UVM with Concurrent Kernels (Multi-Stream) ---\n";
    }

    if (offload_ratio == 0.0f) {
        runSingleKernelTest(N, H, S, trials, true, device_id);
        return;
    }
    
    size_t A_size = (size_t)N * H * sizeof(float);
    size_t B_size = (size_t)H * S * sizeof(float);
    size_t C_size = (size_t)N * S * sizeof(float);
    int N_offload = static_cast<int>(N * offload_ratio);
    int N_resident = N - N_offload;
    size_t A_offload_size = (size_t)N_offload * H * sizeof(float);
    std::cout << "Resident Rows: " << N_resident << ", Offloaded Rows: " << N_offload << std::endl;
    
    float *A, *B, *C;
    CHECK_CUDA(hipMallocManaged(&A, A_size));
    CHECK_CUDA(hipMallocManaged(&B, B_size));
    CHECK_CUDA(hipMallocManaged(&C, C_size));
    init_matrices_on_gpu(A, B, N, H, S);

    std::vector<float> h_A_offload_copy((size_t)N_offload * H);
    float* A_offload_ptr = A + (size_t)N_resident * H;
    CHECK_CUDA(hipMemcpy(h_A_offload_copy.data(), A_offload_ptr, A_offload_size, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipDeviceSynchronize());

    hipStream_t streamCompute, streamTransfer;
    hipEvent_t prefetchDoneEvent;
    CHECK_CUDA(hipStreamCreate(&streamCompute));
    CHECK_CUDA(hipStreamCreate(&streamTransfer));
    CHECK_CUDA(hipEventCreate(&prefetchDoneEvent));
    
    CHECK_CUDA(hipMemAdvise(A, (size_t)N_resident * H * sizeof(float), hipMemAdviseSetPreferredLocation, device_id));
    CHECK_CUDA(hipMemAdvise(A_offload_ptr, A_offload_size, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    CHECK_CUDA(hipMemPrefetchAsync(A, A_size, device_id, streamCompute)); 
    CHECK_CUDA(hipMemPrefetchAsync(A_offload_ptr, A_offload_size, hipCpuDeviceId, streamCompute));
    CHECK_CUDA(hipStreamSynchronize(streamCompute));

    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    CHECK_CUDA(hipGraphCreate(&graph, 0));

    hipEvent_t start_k1, stop_k1, start_k2, stop_k2;
    CHECK_CUDA(hipEventCreate(&start_k1)); CHECK_CUDA(hipEventCreate(&stop_k1));
    CHECK_CUDA(hipEventCreate(&start_k2)); CHECK_CUDA(hipEventCreate(&stop_k2));

    hipKernelNodeParams kernel1_params = {0};
    kernel1_params.func = (void*)matMulKernel;
    kernel1_params.gridDim = calculate_grid_dims(S, N_resident);
    kernel1_params.blockDim = dim3(TILE_DIM, TILE_DIM);
    void *kernel1_args[] = {&C, &A, &B, &N, &H, &S, new int(0), new int(N_resident)};
    kernel1_params.kernelParams = kernel1_args;
    
    hipGraphNode_t start_k1_node, kernel1_node, stop_k1_node;
    CHECK_CUDA(hipGraphAddEventRecordNode(&start_k1_node, graph, nullptr, 0, start_k1));
    CHECK_CUDA(hipGraphAddKernelNode(&kernel1_node, graph, &start_k1_node, 1, &kernel1_params));
    CHECK_CUDA(hipGraphAddEventRecordNode(&stop_k1_node, graph, &kernel1_node, 1, stop_k1));

    hipKernelNodeParams kernel2_params = {0};
    kernel2_params.func = (void*)matMulKernel;
    kernel2_params.gridDim = calculate_grid_dims(S, N_offload);
    kernel2_params.blockDim = dim3(TILE_DIM, TILE_DIM);
    void *kernel2_args[] = {&C, &A, &B, &N, &H, &S, new int(N_resident), new int(N_offload)};
    kernel2_params.kernelParams = kernel2_args;
    
    hipGraphNode_t start_k2_node, kernel2_node, stop_k2_node;
    CHECK_CUDA(hipGraphAddEventRecordNode(&start_k2_node, graph, nullptr, 0, start_k2));
    
    std::vector<hipGraphNode_t> k2_deps;
    k2_deps.push_back(start_k2_node);
    if (SERIALIZE_KERNELS) {
        k2_deps.push_back(stop_k1_node); 
    }
    CHECK_CUDA(hipGraphAddKernelNode(&kernel2_node, graph, k2_deps.data(), k2_deps.size(), &kernel2_params));
    CHECK_CUDA(hipGraphAddEventRecordNode(&stop_k2_node, graph, &kernel2_node, 1, stop_k2));
    
    CHECK_CUDA(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));
    
    const int WARMUP_COUNT = 5;
    std::cout << "Performing " << WARMUP_COUNT << " warm-up runs... " << std::flush;
    for (int i = 0; i < WARMUP_COUNT; ++i) {
        CHECK_CUDA(hipMemcpy(A_offload_ptr, h_A_offload_copy.data(), A_offload_size, hipMemcpyHostToHost));
        CHECK_CUDA(hipMemPrefetchAsync(A_offload_ptr, A_offload_size, device_id, streamTransfer));
        CHECK_CUDA(hipEventRecord(prefetchDoneEvent, streamTransfer));
        CHECK_CUDA(hipGraphLaunch(graph_exec, streamCompute));
        CHECK_CUDA(hipStreamWaitEvent(streamCompute, prefetchDoneEvent));
    }
    CHECK_CUDA(hipDeviceSynchronize());
    std::cout << "Done.\n";
    
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    std::vector<double> total_times, k1_times, k2_times;

    for (int i = 0; i < trials; ++i) {
        CHECK_CUDA(hipMemcpy(A_offload_ptr, h_A_offload_copy.data(), A_offload_size, hipMemcpyHostToHost));
        CHECK_CUDA(hipEventRecord(start, streamCompute));
        CHECK_CUDA(hipMemPrefetchAsync(A_offload_ptr, A_offload_size, device_id, streamTransfer));
        CHECK_CUDA(hipEventRecord(prefetchDoneEvent, streamTransfer));
        CHECK_CUDA(hipGraphLaunch(graph_exec, streamCompute));
        CHECK_CUDA(hipStreamWaitEvent(streamCompute, prefetchDoneEvent));
        CHECK_CUDA(hipEventRecord(stop, streamCompute));
        CHECK_CUDA(hipStreamSynchronize(streamCompute));
        
        float ms_total, ms_k1, ms_k2;
        CHECK_CUDA(hipEventElapsedTime(&ms_total, start, stop));
        CHECK_CUDA(hipEventElapsedTime(&ms_k1, start_k1, stop_k1));
        CHECK_CUDA(hipEventElapsedTime(&ms_k2, start_k2, stop_k2));
        total_times.push_back(ms_total);
        k1_times.push_back(ms_k1);
        k2_times.push_back(ms_k2);
    }

    auto avg = [](const std::vector<double>& v) { return std::accumulate(v.begin(), v.end(), 0.0) / v.size(); };
    std::cout << "\n--- Timings (avg over " << trials << " trials) ---\n";
    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Compute (Resident Data):    " << std::setw(8) << avg(k1_times) << " ms\n";
    std::cout << "Compute (Offloaded Data):   " << std::setw(8) << avg(k2_times) << " ms\n";
    std::cout << "--------------------------------------\n";
    std::cout << "Total Overlapped Time:      " << std::setw(8) << avg(total_times) << " ms\n";
    verify_result_gpu(A, B, C, N, H, S);
    delete (int*)kernel1_args[6]; delete (int*)kernel1_args[7];
    delete (int*)kernel2_args[6]; delete (int*)kernel2_args[7];
    CHECK_CUDA(hipGraphExecDestroy(graph_exec)); CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipEventDestroy(start)); CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipEventDestroy(start_k1)); CHECK_CUDA(hipEventDestroy(stop_k1));
    CHECK_CUDA(hipEventDestroy(start_k2)); CHECK_CUDA(hipEventDestroy(stop_k2));
    CHECK_CUDA(hipEventDestroy(prefetchDoneEvent));
    CHECK_CUDA(hipFree(A)); CHECK_CUDA(hipFree(B)); CHECK_CUDA(hipFree(C));
    CHECK_CUDA(hipStreamDestroy(streamCompute)); CHECK_CUDA(hipStreamDestroy(streamTransfer));
}

// **REWRITTEN:** This is now the primary function for the --extend flag, handling all cases (0, 1, and partial).
void runBandwidthExtensionTest(int N, int H, int S, float offload_ratio, int trials, int device_id, int interleave_ratio) {
    std::cout << "\n--- Running Case 3: Legacy Zero-Copy Test (Interleaved Scratchpad Kernel) ---\n";
    
    // Ensure row counts are aligned to TILE_DIM for simplicity
    if (N % TILE_DIM != 0) {
        int old_N = N;
        N = (N / TILE_DIM) * TILE_DIM;
        std::cout << "Adjusting N from " << old_N << " to " << N << " to be a multiple of TILE_DIM (" << TILE_DIM << ")\n";
    }

    int N_offload = static_cast<int>(round(N * offload_ratio));
    N_offload = (N_offload / TILE_DIM) * TILE_DIM;
    if (offload_ratio > 0.999f) N_offload = N;
    int N_resident = N - N_offload;

    size_t A_resident_size = (size_t)N_resident * H * sizeof(float);
    size_t A_offload_size = (size_t)N_offload * H * sizeof(float);
    size_t A_full_size = (size_t)N * H * sizeof(float);
    size_t B_size = (size_t)H * S * sizeof(float);
    size_t C_size = (size_t)N * S * sizeof(float);

    float *d_A_resident = nullptr, *d_B = nullptr, *d_C = nullptr;
    float *h_A_offload = nullptr;
    float *d_A_offload_mapped = nullptr;

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    
    CHECK_CUDA(hipMalloc(&d_B, B_size));
    CHECK_CUDA(hipMalloc(&d_C, C_size));

    if (N_resident > 0) CHECK_CUDA(hipMalloc(&d_A_resident, A_resident_size));
    if (N_offload > 0) {
        CHECK_CUDA(hipHostAlloc(&h_A_offload, A_offload_size, hipHostMallocMapped));
        CHECK_CUDA(hipHostGetDevicePointer(&d_A_offload_mapped, h_A_offload, 0));
    }
    
    std::cout << "Initializing resident data on GPU and offloaded data on CPU..." << std::flush;
    init_matrices_on_gpu(nullptr, d_B, 0, H, S);
    
    if (N_resident > 0) {
        float* d_A_res_temp;
        CHECK_CUDA(hipMalloc(&d_A_res_temp, A_resident_size));
        init_matrices_on_gpu(d_A_res_temp, nullptr, N_resident, H, 0);
        CHECK_CUDA(hipMemcpy(d_A_resident, d_A_res_temp, A_resident_size, hipMemcpyDeviceToDevice));
        CHECK_CUDA(hipFree(d_A_res_temp));
    }
    if (N_offload > 0) {
        for (size_t i = 0; i < (size_t)N_offload * H; ++i) h_A_offload[i] = (float)rand() / RAND_MAX;
    }
    
    std::cout << "Done.\n";
    std::cout << "Resident Rows: " << N_resident << ", Offloaded Rows: " << N_offload << "\n";
    
    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    CHECK_CUDA(hipGraphCreate(&graph, 0));

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    hipGraphNode_t start_node, kernel_node, stop_node;
    CHECK_CUDA(hipGraphAddEventRecordNode(&start_node, graph, nullptr, 0, start));

    hipKernelNodeParams kernel_params = {0};
    kernel_params.func = (void*)interleavedScratchpadMatMulKernel;
    kernel_params.gridDim = calculate_grid_dims(S, N);
    kernel_params.blockDim = dim3(TILE_DIM, TILE_DIM);

    void *kernel_args[] = {&d_C, &d_A_resident, &d_A_offload_mapped, &d_B, &N, &H, &S, &N_resident, &interleave_ratio};
    kernel_params.kernelParams = kernel_args;

    CHECK_CUDA(hipGraphAddKernelNode(&kernel_node, graph, &start_node, 1, &kernel_params));
    CHECK_CUDA(hipGraphAddEventRecordNode(&stop_node, graph, &kernel_node, 1, stop));
    CHECK_CUDA(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

    const int WARMUP_COUNT = 5;
    std::cout << "Performing " << WARMUP_COUNT << " warm-up runs... " << std::flush;
    for (int i = 0; i < WARMUP_COUNT; ++i) CHECK_CUDA(hipGraphLaunch(graph_exec, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));
    std::cout << "Done.\n";

    std::vector<double> total_times;
    for (int i = 0; i < trials; ++i) {
        CHECK_CUDA(hipGraphLaunch(graph_exec, stream));
        CHECK_CUDA(hipStreamSynchronize(stream));
        
        float ms_total;
        CHECK_CUDA(hipEventElapsedTime(&ms_total, start, stop));
        total_times.push_back(ms_total);
    }

    auto avg = [](const std::vector<double>& v) { return std::accumulate(v.begin(), v.end(), 0.0f) / v.size(); };
    
    double avg_time_ms = avg(total_times);
    double effective_bandwidth = (A_full_size + B_size) / (avg_time_ms * 1e6);
    
    std::cout << "\n--- Timings (avg over " << trials << " trials) ---\n";
    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Effective Blended BW (GB/s): " << std::setw(8) << effective_bandwidth << "\n";
    std::cout << "Total Kernel Time:           " << std::setw(8) << avg_time_ms << " ms\n";
    
    // **RESTORED:** Foolproof verification by reconstructing a simple, contiguous matrix A.
    float* d_A_full_temp_verify;
    CHECK_CUDA(hipMalloc(&d_A_full_temp_verify, (size_t)N * H * sizeof(float)));
    if (N_resident > 0) {
        CHECK_CUDA(hipMemcpy(d_A_full_temp_verify, d_A_resident, A_resident_size, hipMemcpyDeviceToDevice));
    }
    if (N_offload > 0) {
        CHECK_CUDA(hipMemcpy(d_A_full_temp_verify + (size_t)N_resident * H, h_A_offload, A_offload_size, hipMemcpyHostToDevice));
    }

    verify_result_gpu(d_A_full_temp_verify, d_B, d_C, N, H, S);
    
    CHECK_CUDA(hipGraphExecDestroy(graph_exec));
    CHECK_CUDA(hipGraphDestroy(graph));
    CHECK_CUDA(hipEventDestroy(start)); 
    CHECK_CUDA(hipEventDestroy(stop));
    if (d_A_resident) CHECK_CUDA(hipFree(d_A_resident));
    if (h_A_offload) CHECK_CUDA(hipHostFree(h_A_offload));
    CHECK_CUDA(hipFree(d_A_full_temp_verify));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipStreamDestroy(stream));
}

// Prints the command line usage instructions
void print_usage(const char* prog_name) {
    std::cerr << "\nUsage: " << prog_name << " [options]\n\n";
    std::cerr << "Options:\n";
    std::cerr << "  -h, --help                         Show this help message and exit.\n";
    std::cerr << "  --nvlink                           Use NVLink for D2D transfer (Case 1, requires 2+ GPUs).\n";
    std::cerr << "  --uvm                              Use UVM with Prefetch (Case 2).\n";
    std::cerr << "  --extend                           Use Legacy Zero-Copy Test (Interleaved Kernel, Case 3).\n";
    std::cerr << "  --interleave <X>                   Set X-to-1 resident/offload interleave ratio for --extend mode. (Default: 9)\n";
    std::cerr << "  -N, --N, --rows <int>              Number of rows for matrix A. (Default: 1000000)\n";
    std::cerr << "  -H, --H, --hidden_dim <int>        Number of columns for A / rows for B. (Default: 1024)\n";
    std::cerr << "  -S, --S, --cols <int>              Number of columns for matrix B. (Default: 1)\n";
    std::cerr << "  -r, --ratio, --offload_ratio <f>   Fraction of matrix A to offload (0.0 to 1.0). (Default: 0.1)\n";
    std::cerr << "  -t, --trials <int>                 Number of timed trials to run. (Default: 1000)\n";
    std::cerr << "  -d, --device <id>                  ID of the GPU device to use. (Default: 0)\n\n";
    std::cerr << "Note: Default is Explicit Overlap (PCIe). Test modes (--nvlink, --uvm, --extend) are mutually exclusive.\n";
}

// Simple command line parser with error handling
bool parse_args(int argc, char** argv, int& N, int& H, int& S, float& ratio, int& trials, bool& use_uvm, bool& use_extension, bool& use_nvlink, int& device_id, int& interleave_ratio) {
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "--help" || arg == "-h") {
            print_usage(argv[0]);
            return false;
        } else if (arg == "--nvlink") {
            use_nvlink = true;
        } else if (arg == "--uvm") {
            use_uvm = true;
        } else if (arg == "--extend") {
            use_extension = true;
        } else if (arg == "--interleave" && i + 1 < argc) {
            interleave_ratio = std::stoi(argv[++i]);
        } else if ((arg == "--N" || arg == "-N" || arg == "--rows") && i + 1 < argc) {
            N = std::stoi(argv[++i]);
        } else if ((arg == "--H" || arg == "-H" || arg == "--hidden_dim") && i + 1 < argc) {
            H = std::stoi(argv[++i]);
        } else if ((arg == "--S" || arg == "-S" || arg == "--cols") && i + 1 < argc) {
            S = std::stoi(argv[++i]);
        } else if ((arg == "--offload_ratio" || arg == "-r" || arg == "--ratio") && i + 1 < argc) {
            ratio = std::stof(argv[++i]);
        } else if ((arg == "--trials" || arg == "-t") && i + 1 < argc) {
            trials = std::stoi(argv[++i]);
        } else if ((arg == "--device" || arg == "-d") && i + 1 < argc) {
            device_id = std::stoi(argv[++i]);
        } else {
            std::cerr << "Error: Unknown or invalid argument: " << arg << std::endl;
            print_usage(argv[0]);
            return false;
        }
    }

    if ((use_uvm && use_extension) || (use_uvm && use_nvlink) || (use_extension && use_nvlink)) {
        std::cerr << "Error: --uvm, --extend, and --nvlink flags are mutually exclusive." << std::endl;
        print_usage(argv[0]);
        return false;
    }
    if (N <= 0 || H <= 0 || S <= 0 || trials <= 0 || interleave_ratio <= 0) {
        std::cerr << "Error: Matrix dimensions, trial count, and interleave ratio must be positive." << std::endl;
        print_usage(argv[0]);
        return false;
    }
    if (ratio < 0.0f || ratio > 1.0f) {
        std::cerr << "Error: Offload ratio must be in the range [0.0, 1.0]." << std::endl;
        print_usage(argv[0]);
        return false;
    }
     if (device_id < 0) {
        std::cerr << "Error: Device ID must be a non-negative integer." << std::endl;
        return false;
    }
    int device_count;
    CHECK_CUDA(hipGetDeviceCount(&device_count));
    if (device_id >= device_count) {
        std::cerr << "Error: Device ID " << device_id << " is invalid. Only " << device_count << " devices found on this system." << std::endl;
        return false;
    }
    if (use_nvlink && device_count < 2) {
        std::cerr << "Error: --nvlink mode requires at least 2 GPUs." << std::endl;
        return false;
    }
    return true; // Success
}

int main(int argc, char** argv)
{
    // Default values
    int N = 1000000, H = 1024, S = 1;
    float offload_ratio = 0.1f;
    int trials = 1000;
    bool use_uvm = false;
    bool use_extension = false;
    bool use_nvlink = false;
    int device_id = 0;
    int interleave_ratio = 9; // Default X for X-to-1 interleaving

    if (!parse_args(argc, argv, N, H, S, offload_ratio, trials, use_uvm, use_extension, use_nvlink, device_id, interleave_ratio)) {
        return 1;
    }

    CHECK_CUDA(hipSetDevice(device_id));

    std::cout << "Configuration:\n";
    std::string mode = "Explicit Overlap with PCIe (Case 1)";
    if (use_nvlink) mode = "Explicit Overlap with NVLink (Case 1)";
    if (use_uvm) mode = "UVM with Prefetch (Case 2)";
    if (use_extension) mode = "Legacy Zero-Copy (Interleaved Kernel, Case 3)";


    std::cout << "  Mode:          " << mode << "\n";
    std::cout << "  Device ID:     " << device_id << "\n";
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, device_id));
    std::cout << "  Device Name:   " << prop.name << "\n";
    std::cout << "  Matrix A:      " << N << " x " << H << "\n";
    std::cout << "  Matrix B:      " << H << " x " << S << "\n";
    std::cout << "  Offload Ratio: " << offload_ratio * 100 << "%\n";
    if (use_extension && offload_ratio > 0.0f && offload_ratio < 1.0f) {
        std::cout << "  Interleave Ratio: " << interleave_ratio << ":1\n";
    }
    std::cout << "  Trials:        " << trials << "\n";

    if (use_extension) {
        runBandwidthExtensionTest(N, H, S, offload_ratio, trials, device_id, interleave_ratio);
    } else if (use_uvm) {
        runUvmTest(N, H, S, offload_ratio, trials, device_id);
    } else {
        runExplicitOverlapTest(N, H, S, offload_ratio, trials, use_nvlink, device_id);
    }

    return 0;
}

