
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <numeric> // For std::accumulate

// Error checking macro
#define CHECK(call)                                                            \
    do {                                                                       \
        const hipError_t error_code = call;                                   \
        if (error_code != hipSuccess) {                                       \
            printf("CUDA Error:\n");                                           \
            printf("    File:       %s\n", __FILE__);                           \
            printf("    Line:       %d\n", __LINE__);                           \
            printf("    Error code: %d\n", error_code);                         \
            printf("    Error text: %s\n", hipGetErrorString(error_code));     \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

// A compute-intensive kernel for testing timing
__global__ void intensive_kernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ITERATIONS = 2500;

    if (idx < n) {
        float val = (float)data[idx];
        for (int i = 0; i < ITERATIONS; ++i) {
            val = sinf(val) * 0.5f + cosf(val) * 0.8f;
        }
        data[idx] = (int)val;
    }
}

int main() {
    // 1. Setup
    int n = 4 * 1024 * 1024;
    size_t bytes = n * sizeof(int);

    std::vector<int> h_data(n);
    std::vector<int> h_result(n);
    for (int i = 0; i < n; ++i) {
        h_data[i] = i;
    }

    int *d_data;
    CHECK(hipMalloc(&d_data, bytes));
    CHECK(hipMemcpy(d_data, h_data.data(), bytes, hipMemcpyHostToDevice));

    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));

    // NEW: Create event pairs for each kernel and for the total graph
    hipEvent_t start_A, stop_A, start_B, stop_B, start_C, stop_C;
    hipEvent_t start_graph, stop_graph;
    CHECK(hipEventCreate(&start_A)); CHECK(hipEventCreate(&stop_A));
    CHECK(hipEventCreate(&start_B)); CHECK(hipEventCreate(&stop_B));
    CHECK(hipEventCreate(&start_C)); CHECK(hipEventCreate(&stop_C));
    CHECK(hipEventCreate(&start_graph)); CHECK(hipEventCreate(&stop_graph));

    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    CHECK(hipGraphCreate(&graph, 0));

    // Define the kernel launch parameters (reused for all 3 kernels)
    hipKernelNodeParams kernel_params = {0};
    kernel_params.func = (void*)intensive_kernel;
    kernel_params.gridDim = dim3((n + 255) / 256, 1, 1);
    kernel_params.blockDim = dim3(256, 1, 1);
    void *kernel_args[] = {&d_data, &n};
    kernel_params.kernelParams = kernel_args;

    // --- NEW: Build Graph with Event Nodes Around Each Kernel ---
    hipGraphNode_t node_A, node_B, node_C;
    hipGraphNode_t event_start_A, event_stop_A, event_start_B, event_stop_B, event_start_C, event_stop_C;

    // Chain 1: Kernel A
    CHECK(hipGraphAddEventRecordNode(&event_start_A, graph, nullptr, 0, start_A));
    CHECK(hipGraphAddKernelNode(&node_A, graph, &event_start_A, 1, &kernel_params));
    CHECK(hipGraphAddEventRecordNode(&event_stop_A, graph, &node_A, 1, stop_A));

    // Chain 2: Kernel B (depends on Kernel A finishing)
    CHECK(hipGraphAddEventRecordNode(&event_start_B, graph, &event_stop_A, 1, start_B));
    CHECK(hipGraphAddKernelNode(&node_B, graph, &event_start_B, 1, &kernel_params));
    CHECK(hipGraphAddEventRecordNode(&event_stop_B, graph, &node_B, 1, stop_B));

    // Chain 3: Kernel C (depends on Kernel B finishing)
    CHECK(hipGraphAddEventRecordNode(&event_start_C, graph, &event_stop_B, 1, start_C));
    CHECK(hipGraphAddKernelNode(&node_C, graph, &event_start_C, 1, &kernel_params));
    CHECK(hipGraphAddEventRecordNode(&event_stop_C, graph, &node_C, 1, stop_C));

    // --- Instantiate, Launch, and Time ---
    CHECK(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

    // NEW: Record events on the stream around the graph launch
    CHECK(hipEventRecord(start_graph, stream));
    CHECK(hipGraphLaunch(graph_exec, stream));
    CHECK(hipEventRecord(stop_graph, stream));

    CHECK(hipStreamSynchronize(stream));

    // --- NEW: Calculate and Report All Timings ---
    float time_A = 0, time_B = 0, time_C = 0, time_graph = 0;
    CHECK(hipEventElapsedTime(&time_A, start_A, stop_A));
    CHECK(hipEventElapsedTime(&time_B, start_B, stop_B));
    CHECK(hipEventElapsedTime(&time_C, start_C, stop_C));
    CHECK(hipEventElapsedTime(&time_graph, start_graph, stop_graph));

    CHECK(hipMemcpy(h_result.data(), d_data, bytes, hipMemcpyDeviceToHost));

    std::cout.precision(4);
    std::cout << std::fixed;
    std::cout << "--- Timing Results ---" << std::endl;
    std::cout << "Kernel A Execution Time: \t" << time_A << " ms" << std::endl;
    std::cout << "Kernel B Execution Time: \t" << time_B << " ms" << std::endl;
    std::cout << "Kernel C Execution Time: \t" << time_C << " ms" << std::endl;
    std::cout << "----------------------------------" << std::endl;
    float sum_kernels = time_A + time_B + time_C;
    std::cout << "Sum of Kernel Times: \t\t" << sum_kernels << " ms" << std::endl;
    std::cout << "Total Graph Execution Time: \t" << time_graph << " ms" << std::endl;

    // --- Cleanup ---
    CHECK(hipEventDestroy(start_A)); CHECK(hipEventDestroy(stop_A));
    CHECK(hipEventDestroy(start_B)); CHECK(hipEventDestroy(stop_B));
    CHECK(hipEventDestroy(start_C)); CHECK(hipEventDestroy(stop_C));
    CHECK(hipEventDestroy(start_graph)); CHECK(hipEventDestroy(stop_graph));
    CHECK(hipGraphExecDestroy(graph_exec));
    CHECK(hipGraphDestroy(graph));
    CHECK(hipStreamDestroy(stream));
    CHECK(hipFree(d_data));

    return 0;
}